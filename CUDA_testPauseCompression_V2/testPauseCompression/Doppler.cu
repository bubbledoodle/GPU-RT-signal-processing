#include ""
#include "ExecutionRD.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/device_functions.h>

hipComplex *d_doppler;
void FFT(Complex *d_input);

void Doppler(Complex *h_in)
{
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	{
		int size_doppler = sizeof(Complex)* BATCH * LENGTH;
		hipMalloc((void**)&d_doppler, size_doppler);
		hipMemcpy(d_doppler, h_in, size_doppler, hipMemcpyHostToDevice);
		FFT(d_doppler);
		hipMemcpy(h_in, d_doppler, size_doppler, hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Doppler CUDA runtime is %f sec\n", time / 1e3);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_doppler);
}

void FFT(Complex *d_input)
{
	hipfftHandle plan;

	// parameters
	#define RANK 1
	int n[RANK] = { NX };
	int istride = NX, ostride = NX;
	int idist = 1, odist = 1;
	int *inembed = NULL, *onembed = NULL;

	hipfftPlanMany(&plan, RANK, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, BATCH);
	hipfftExecC2C(plan, (hipfftComplex*)d_input, (hipfftComplex*)d_input, HIPFFT_FORWARD);
	hipDeviceSynchronize();

	hipfftDestroy(plan);
}