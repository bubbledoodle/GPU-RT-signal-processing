#include "hip/hip_runtime.h"
#include "ExecutionRD.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include ""
#include <hipblas.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

hipComplex *d_vecX;
hipComplex *d_ref;
hipComplex *d_out;

//Kernel
void FFT(Complex *d_input1, Complex *d_input2);
void IFFT(Complex *d_input);
void matrixMul(Complex *d_buf, Complex * d_ref, Complex *Oput);

static __global__ void devmatrixMul(hipComplex *X, hipComplex *Y, hipComplex *Oput);

static __device__ __host__ inline hipComplex ComplexMul(hipComplex a, hipComplex b);
static __device__ __host__ inline hipComplex ComplexConjugate(hipComplex a);

void PauseCompression(Complex *h_buf, Complex *h_ref, Complex *Oput, Complex *test)
{
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	{
		hipError_t error;
		int size_d_vecX = sizeof(Complex)* BATCH * LENGTH;
		int size_d_ref = sizeof(Complex)* BATCH * LENGTH;
		int size_d_out = sizeof(Complex)* BATCH * LENGTH;
		hipMalloc((void**)&d_vecX, size_d_vecX);
		hipMalloc((void**)&d_ref, size_d_ref);
		hipMalloc((void**)&d_out, size_d_out);

		error = hipMemcpy(d_vecX, h_buf, size_d_vecX, hipMemcpyHostToDevice);
		error = hipMemcpy(d_ref, h_ref, size_d_ref, hipMemcpyHostToDevice);

		FFT(d_ref, d_vecX);
		matrixMul(d_ref, d_vecX, d_out);
		IFFT(d_out);

		//hipMemcpy(test, d_ref, size_d_ref, hipMemcpyDeviceToHost);
		hipMemcpy(Oput, d_out, size_d_out, hipMemcpyDeviceToHost);
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Pause Compression CUDA runtime is %f sec\n", time / 1e3);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//hipFree
	hipFree(d_vecX);
	hipFree(d_ref);
	hipFree(d_out);
}

void FFT(Complex *d_input1, Complex *d_input2)
{
	hipfftHandle plan;

	// parameters
	#define RANK 1
	int n[RANK] = { NX };
	int istride = NX, ostride = NX;
	int idist = 1, odist = 1;
	int *inembed = NULL, *onembed = NULL;

	hipfftPlanMany(&plan, RANK, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, BATCH);
	hipfftExecC2C(plan, (hipfftComplex*)d_input1, (hipfftComplex*)d_input1, HIPFFT_FORWARD);
	hipfftExecC2C(plan, (hipfftComplex*)d_input2, (hipfftComplex*)d_input2, HIPFFT_FORWARD);
	hipDeviceSynchronize();

	hipfftDestroy(plan);
}

void matrixMul(Complex *d_buf, Complex * d_ref, Complex *Oput)
{
	dim3 dimBlock(1024, 1);
	int dimGrid = BATCH * LENGTH / 1024;
	devmatrixMul << <dimGrid, dimBlock >> >(d_ref, d_buf, Oput);
}

static __global__ void devmatrixMul(hipComplex *X, hipComplex *Y, hipComplex *Oput)
{
	//block index
	int bx = blockIdx.x;

	//threads index
	int tx = threadIdx.x;

	int xBegin = bx * dimBlock_x;

	Oput[xBegin + tx] = ComplexMul(X[xBegin + tx], ComplexConjugate(Y[xBegin + tx]));
	__syncthreads();
}

void IFFT(Complex *d_input)
{
	hipfftHandle plan;
	// parameters
	#define RANK 1
	int n[RANK] = { NX };
	int istride = NX, ostride = NX;
	int idist = 1, odist = 1;
	int *inembed = NULL, *onembed = NULL;

	hipfftPlanMany(&plan, RANK, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, BATCH);
	hipfftExecC2C(plan, (hipfftComplex*)d_input, (hipfftComplex*)d_input, HIPFFT_BACKWARD);
	hipDeviceSynchronize();

	hipfftDestroy(plan);
}
static __device__ __host__ inline hipComplex ComplexMul(hipComplex a, hipComplex b)
{
	hipComplex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

static __device__ __host__ inline hipComplex ComplexConjugate(hipComplex a)
{
	hipComplex b;
	b.x = a.x;
	b.y = -a.y;
	return b;
}