#include "hip/hip_runtime.h"
// This kernel is for big device memory(bigger than 2 Gigabytes)
#include "ExecutionDBF.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include ""
#include <hipblas.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

void matrixMul(Complex *d_in1, Complex * d_in2, Complex * d_out);

static __device__ __host__ inline hipComplex ComplexMul(hipComplex a, hipComplex b);
static __device__ __host__ inline hipComplex ComplexAdd(hipComplex a, hipComplex b);
static __global__ void devmatrixMul(hipComplex *vecX, hipComplex *w, hipComplex *d_Oput);

void DBF(Complex *echo, Complex *Oput, Complex *w)
{
	hipComplex *d_vecX;
	hipComplex *d_w;
	hipComplex *d_Oput;

	int size_d_vecX = sizeof(Complex)* NX * BATCH * CHANNEL/2;	//array signal
	int size_d_w = sizeof(Complex)* CHANNEL * ANGLE;			//weight
	int size_d_Oput = sizeof(Complex) * NX * ANGLE * BATCH/2;	//store beam

	hipMalloc((void**)&d_vecX, size_d_vecX);
	hipMalloc((void**)&d_Oput, size_d_Oput);
	hipMalloc((void**)&d_w, size_d_w);

	hipMemcpy(d_vecX, echo, size_d_vecX, hipMemcpyHostToDevice);
	hipMemcpy(d_w, w, size_d_w, hipMemcpyHostToDevice);
	matrixMul(d_vecX, d_w, d_Oput);

	hipMemcpy(Oput, d_Oput, size_d_Oput, hipMemcpyDeviceToHost);

	//hipFree
	hipFree(d_vecX);
	hipFree(d_w);
	hipFree(d_Oput);
}

void matrixMul(Complex *d_in1, Complex * d_in2, Complex * d_out)
{
	dim3 dimBlock(32, 31);
	dim3 dimGrid = BATCH * LENGTH / 64;
	devmatrixMul << <dimGrid, dimBlock >> >(d_in1, d_in2, d_out);
}

static __global__ void devmatrixMul(hipComplex *vecX, hipComplex *w, hipComplex *out)
{
	hipComplex Csub = { 0, 0 };
	hipComplex ref = { 0, 0 };
	hipComplex b = { 0, 0 };

	//block index
	int bx = blockIdx.x;

	//threads index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int xBegin = bx * 32;
	
	for (int k = 0; k < CHANNEL; ++k)
	{
		ref = w[k + tx];
		b = vecX[xBegin + tx + k*LENGTH*BATCH / 2];
		Csub = ComplexAdd(Csub, ComplexMul(b, ref));
	}
	out[xBegin + tx + ty * LENGTH*BATCH] = Csub;
	__syncthreads();
}

static __device__ __host__ inline hipComplex ComplexAdd(hipComplex a, hipComplex b)
{
	hipComplex c;
	c.x = a.x + b.x;
	c.y = a.y + b.y;
	return c;
}
static __device__ __host__ inline hipComplex ComplexMul(hipComplex a, hipComplex b)
{
	hipComplex c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}